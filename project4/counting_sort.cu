#include <hip/hip_runtime.h>
#include <stdio.h>

#define nthread 256

__global__ void count_hist(int arr[], int hist[], int size)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < size) atomicAdd(&hist[arr[i]], 1);
}

__global__ void arr_sort(int arr[], int hist[], int max_val)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = threadIdx.x;
	__shared__ int s_idx[nthread];

	if(i < max_val){
		s_idx[j] = hist[i];
		
		__syncthreads();

		int s, cnt;
		if(i==0 && j==0) s=0;
		else if(i!=0 && j==0) s=hist[i-1];
		else s=s_idx[j-1];

		cnt = s_idx[j] - s;

		for(int idx = s; idx < s+cnt; idx++){
			arr[idx] = i;
		}
	}

}

__host__ void counting_sort(int arr[], int size, int max_val)
{
   // fill
	int* histogram;
	int* cuda_hist, *cuda_arr;
	histogram = (int *)calloc(sizeof(int), max_val);

	hipMalloc((void**)&cuda_hist, max_val*sizeof(int));
	hipMalloc((void**)&cuda_arr, size*sizeof(int));

	hipMemcpy(cuda_hist, histogram, max_val*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_arr, arr, size*sizeof(int), hipMemcpyHostToDevice);

	count_hist <<< ceil((double)size / nthread), nthread >>> (cuda_arr, cuda_hist, size);


	hipMemcpy(histogram, cuda_hist, max_val*sizeof(int), hipMemcpyDeviceToHost);

//	cudaFree(cuda_hist);
//	cudaFree(cuda_arr);

	int sum = 0;
	for (int i=0; i<max_val; i++)
	{
		sum += histogram[i];
		histogram[i] = sum;
	}

	hipMemcpy(cuda_hist, histogram, max_val*sizeof(int), hipMemcpyHostToDevice);

	arr_sort <<< ceil((double)max_val / nthread), nthread  >>> (cuda_arr, cuda_hist, max_val);

	hipMemcpy(arr, cuda_arr, size*sizeof(int), hipMemcpyDeviceToHost);

	hipFree(cuda_hist);
	hipFree(cuda_arr);
	free(histogram);
}

